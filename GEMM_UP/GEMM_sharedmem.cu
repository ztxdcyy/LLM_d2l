//核函数的具体实现
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <chrono>
#include <hip/hip_runtime.h>
#define BLOCK_SIZE 32

__global__ void matmul_ShareMemory(float *M,float *N,float *P,int width){
    __shared__ float Mds[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Nds[BLOCK_SIZE][BLOCK_SIZE];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int Col = bx * BLOCK_SIZE + tx;
    int Row = by * BLOCK_SIZE + ty;

    int Pervalue = 0;
    //有多少个BLOCK_SIZE，每个循环计算一个块的大小
    for(int i = 0;i < width / BLOCK_SIZE;i++){
        Mds[ty][tx] = M[Row * width + (i * BLOCK_SIZE + tx)];
        Nds[ty][tx] = N[Col + (i * BLOCK_SIZE + ty) * width];
        __syncthreads();        // 确保所有线程都完成了共享内存的写入

        //BLOCK_SIZE相乘
        for(int k = 0;k < BLOCK_SIZE;k++)
            Pervalue += Mds[ty][k] * Nds[k][tx];
        __syncthreads();
    }
    P[Row * width + Col] = Pervalue;
}

// 检查CUDA错误
void checkCudaError(hipError_t err, const char *msg) {
    if (err != hipSuccess) {
        std::cerr << msg << ": " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

int main() {
    const int sizes[] = {512, 1024, 2048, 4096};
    // const int blockSize = 32;               // 为了满足研究warp的性质，设置blocksize为32，因为warp的定义是32个内存连续的线程，统一定义为一个事件，接受指令transaction，而不是一个线程一个事件，这样可以减少指令的数量，提高效率。

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    std::cout << "Max grid size: " 
            << prop.maxGridSize[0] << " x "
            << prop.maxGridSize[1] << " x "
            << prop.maxGridSize[2] << std::endl;
    std::cout << "Max block dimensions: " 
            << prop.maxThreadsDim[0] << " x "
            << prop.maxThreadsDim[1] << " x "
            << prop.maxThreadsDim[2] << std::endl;

    for (int width : sizes) {
        // 分配主机内存
        size_t matrixSize = width * width * sizeof(float);
        float *h_A = new float[width * width];
        float *h_B = new float[width * width];
        float *h_C = new float[width * width];

        // 初始化随机数
        std::srand(std::time(nullptr));
        for (int i = 0; i < width * width; i++) {
            h_A[i] = static_cast<float>(std::rand()) / RAND_MAX;
            h_B[i] = static_cast<float>(std::rand()) / RAND_MAX;
        }

        // 分配设备内存
        float *d_A, *d_B, *d_C;
        checkCudaError(hipMalloc(&d_A, matrixSize), "Failed to allocate d_A");
        checkCudaError(hipMalloc(&d_B, matrixSize), "Failed to allocate d_B");
        checkCudaError(hipMalloc(&d_C, matrixSize), "Failed to allocate d_C");

        // 拷贝数据到设备
        checkCudaError(hipMemcpy(d_A, h_A, matrixSize, hipMemcpyHostToDevice), "Failed to copy A");
        checkCudaError(hipMemcpy(d_B, h_B, matrixSize, hipMemcpyHostToDevice), "Failed to copy B");

        // 配置CUDA核函数，按照blocksize划分grid，使得整个输入矩阵width都能被per element的分配到内存上。
        dim3 block(BLOCK_SIZE, BLOCK_SIZE);
        dim3 grid((width + block.x - 1) / block.x, (width + block.y - 1) / block.y);

        // 执行并计时
        auto start = std::chrono::high_resolution_clock::now();
        matmul_ShareMemory<<<grid, block>>>(d_A, d_B, d_C, width);
        hipDeviceSynchronize();
        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> elapsed = end - start;

        // 检查核函数执行错误
        checkCudaError(hipGetLastError(), "Kernel execution failed");

        // 拷贝结果回主机
        checkCudaError(hipMemcpy(h_C, d_C, matrixSize, hipMemcpyDeviceToHost), "Failed to copy C");

        std::cout << "Matrix multiplication (" << width << "x" << width << ") completed in: " << elapsed.count() << " seconds" << std::endl;

        // 释放资源
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        delete[] h_A;
        delete[] h_B;
        delete[] h_C;
    }
    return 0;
}
